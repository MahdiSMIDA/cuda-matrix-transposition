
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

int transponowanie(){
  clock_t begin = clock();
  int const size(1000);
  static double tablica[size][size];
  static double tab[size][size];
  for(int i=0; i<size;i++){
    for(int j=0; j<size;j++){
      tablica[i][j]=i*size+j+1;
    }
  }
  printf("przed: 1:%f ", tablica[0][1]);
  printf("2:%f \n", tablica[1][0]);
  for(int i=0; i<size;i++){
    for(int j=0; j<size;j++){
      tab[j][i]=tablica[i][j];
    }
  }
  printf("po: 1:%f ", tab[0][1]);
  printf("2:%f \n", tab[1][0]);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("czas CPU: %f \n", elapsed_secs);
  return 1;
}




int main(void){
  transponowanie();
}
